#include <fstream>
#include "file_io.h"
#include "data_manip.h"
#include "neural_net.h"
#include "mat_mult.h"
#include "hip/hip_runtime.h"
#include "matrix.h"
#include <iostream>


int main(void)
{   
    std::string filename = "iris.csv";

    int height = 150, width = 5;
    struct Matrix *dataset = create_matrix(height, width);
    if(!read_data(dataset, filename, false))
    {
        printf("Error reading data\n");
        return -1;
    }
    // print_matrix(dataset);
    // printf("\n");
    // printf("File read successfully\n");
      normalize_data(dataset);

    int train_size = 1;
    
    struct Matrix *train, *test;
    train = create_matrix(train_size, width);
    test = create_matrix(height - train_size, width);
    split_data(dataset, train, test);
    // print_matrix(train);
    // printf("\n");
    // print_matrix(test);
    // float *ground_truth = (float *) malloc(3* train_size * sizeof(float));
    struct Matrix *ground_truth = create_matrix(train_size, 3);
    create_ground_truth(train, ground_truth);
    // print_matrix(ground_truth);
    // // validate_split(train, test, height, width, train_size);
    print_matrix(train);
  
    printf("\n");
    print_matrix(train);
    printf("\n");
    print_matrix(ground_truth);

    int topology[] = {4, 3};
    NeuralNetwork net(topology, 2);
    net.train(train,ground_truth,10000,.001);

    // printf("\n%f\n", dataset[0]);
    // float *inputs, *weights, *outputs;
    // int a = 2, b = 3,c = 4;
    // inputs = (float *) malloc(a * b * sizeof(float));
    // weights = (float *) malloc((b+1) * c * sizeof(float));
    // outputs = (float *) malloc(a * c * sizeof(float));

    // for(int i = 0; i < a * b; i++)
    // {
    //     inputs[i] = i;
    // }
    // for(int i = 0; i < (b+1) * c; i++)
    // {
    //     weights[i] = i;
    // }
    // for(int i = 0; i < a * c; i++)
    // {
    //     outputs[i] = 0;
    // }
    // float *d_inputs, *d_weights, *d_outputs;
    // hipMalloc(&d_inputs, a * b * sizeof(float));
    // hipMalloc(&d_weights, (b+1) * c * sizeof(float));
    // hipMalloc(&d_outputs, a * c * sizeof(float));

    // hipMemcpy(d_inputs, inputs,  a * b * sizeof(float), hipMemcpyHostToDevice);
    // hipMemcpy(d_weights, weights,  (b+1) * c * sizeof(float), hipMemcpyHostToDevice);
    // // printf("\n%f\n", d_weights[0]);
    
    // nn_mat_mul(d_inputs, d_weights, d_outputs, a, b, c);
    // std::cout << "Inputs: " << std::endl;
    // hipMemcpy(outputs,d_outputs, a * c * sizeof(float), hipMemcpyDeviceToHost);

    // for (int y = 0; y < a; y++)
    // {
    //     for (int x = 0; x < b; x++)
    //     {
    //         printf("%f ", inputs[y * a + x]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");

    // for (int y = 0; y < b+1; y++)
    // {
    //     for (int x = 0; x < c; x++)
    //     {
    //         printf("%f ", weights[y * (b+1) + x]);
    //     }
    //     printf("\n");
    // }
    // printf("\n");

    // for (int y = 0; y < a; y++) {
    //     for (int x = 0; x < c; x++) {
    //         printf("%f ", outputs[y * c + x]);
    //     }
    //     printf("\n");
    // }
    return 0;
}
